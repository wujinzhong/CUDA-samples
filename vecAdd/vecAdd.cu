#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void vecAdd( float* A, float* B, float* C, int N )
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i<N )
        C[i] = A[i] + B[i];
}

int main(void)
{
    srand(time(0));
    int N = 1024*1024;
    size_t sz = N*sizeof(float);
    float* hA = (float*)malloc(sz);
    float* hB = (float*)malloc(sz);
    float* hC = (float*)malloc(sz);
    for( int i=0; i<N; i++ )
    {
        hA[i] = rand() & 0xff;
        hB[i] = rand() & 0xff;
        hC[i] = 0;
    }

    float* dA, *dB, *dC;
    hipMalloc( (void **)&dA, sz );
    hipMalloc( (void **)&dB, sz );
    hipMalloc( (void **)&dC, sz );

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync( dA, hA, sz, hipMemcpyHostToDevice, stream );
    hipMemcpyAsync( dB, hB, sz, hipMemcpyHostToDevice, stream );
    vecAdd<<<N/1024, 1024, 0, stream>>>(dA, dB, dC, N);

    hipMemcpyAsync( hC, dC, sz, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize( stream );

    for( int i=0; i<10; i++ )
    {
        printf("%8f + %8f = %8f\n", hA[i], hB[i], hC[i]);
    }

    free(hA);
    free(hB);
    free(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipStreamDestroy(stream);

    return 0;
}
